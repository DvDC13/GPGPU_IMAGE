#include "hip/hip_runtime.h"
#include <algorithm>
#include <filesystem>

#include "wrapper_gpu.cuh"
#include "similarityMeasuresC.cuh"
#include "featuresExtractionT.cuh"
#include "similarityMeasuresT.cuh"
#include "choquet.cuh"

#include "image.cuh"

hipStream_t stream1;
hipStream_t stream2;

uint8_t* getBitVector(shared_image image)
{
    int width = image->get_width();
    int height = image->get_height();
    int size = width * height;

    // Allocate device memory
    Pixel* deviceImageData;
    uint8_t* deviceBitVectorData;
    cudaXMalloc((void**)&deviceImageData, size * sizeof(Pixel));
    cudaXMalloc((void**)&deviceBitVectorData, size * sizeof(uint8_t));

    // Copy image data from host to device
    cudaXMemcpy(deviceImageData, image->get_data().data(), size * sizeof(Pixel), hipMemcpyHostToDevice);

    // Launch the kernel
    dim3 blockSize(32, 32);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    calculateBitVectorBackground<<<gridSize, blockSize>>>(deviceImageData, deviceBitVectorData, width, height);
    hipDeviceSynchronize();

    return deviceBitVectorData;
}

int main(int argc, char** argv)
{
    if (argc != 2)
    {
        std::cout << "Usage: " << argv[0] << " <path_to_dataset>" << std::endl;
        return EXIT_FAILURE;
    }

    std::vector<std::string> files;
    std::string path = std::string(argv[1]);

    std::vector<shared_image> images;

    for (const auto& entry : std::filesystem::directory_iterator(path))
        files.push_back(entry.path());

    std::sort(files.begin(), files.end());

    files.reserve(files.size() - 1);


    shared_image background = load_png(files[0]);

    size_t height = background->get_height();
    size_t width = background->get_width();

    // for (auto it = files.begin() + 1; it != files.end(); it++)
    //     images.push_back(load_png(*it));

    size_t memory_usage = width * height * ((sizeof(Pixel)) + sizeof(uint8_t) + sizeof(float) + sizeof(Bit) + sizeof(std::array<float, 2>));
    size_t memory_usage_bg = width * height * (sizeof(uint8_t) + sizeof(Pixel));

    size_t maximum_global_memory = 0;
    hipMemGetInfo(&maximum_global_memory, nullptr);
    size_t max_batch_size = std::floor((maximum_global_memory - memory_usage_bg) / memory_usage);
    size_t batch_size = std::min(max_batch_size, files.size());

    std::vector<Pixel *> batches;
    for (size_t batch_num = 0; batch_num < std::ceil(float(files.size()) /
                float(batch_size)); batch_num++)
    {
        auto start_iter =  files.begin() + batch_num * batch_size;
        auto end_iter = files.begin() +  std::min(
                            (batch_num + 1) * batch_size,
                            files.size()
                            );
        std::vector<std::string> subvect = std::vector<std::string>(start_iter,
                end_iter);
        Pixel* batch = load_image_batch(subvect);
        batches.push_back(batch);
    }

    std::cout << "Height: " << height << std::endl;
    std::cout << "Width: " << width << std::endl;
    std::cout << "Memory usage: " << memory_usage << std::endl;
    std::cout << "Memory usage background: " << memory_usage_bg << std::endl;
    std::cout << "Maximum global memory: " << maximum_global_memory << std::endl;
    std::cout << "Maximum batch size: " << max_batch_size << std::endl;
    std::cout << "Batch size: " << batch_size << std::endl;

    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    hipStream_t stream1;
    hipStream_t stream2;

    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    uint8_t* backgroundBitVector = getBitVector(background);

    Pixel* backgroundData;
    cudaXMalloc((void**)&backgroundData, width * height * sizeof(Pixel));
    cudaXMemcpy(backgroundData, background->get_data().data(), width * height * sizeof(Pixel), hipMemcpyHostToDevice);


    //void cudaXMalloc3D(void** devPtr, size_t elem_size, size_t* pitch, size_t w, size_t h, size_t d)
    
    hipPitchedPtr imagesPtr = { 0 };
    // cudaXMalloc((void**)&imagesData, width * height * batch_size * sizeof(Pixel));
    cudaXMalloc3D(&imagesPtr, sizeof(Pixel), width, height, batch_size);
    Pixel* imagesData = (Pixel*) imagesPtr.ptr;
    size_t imagePitch = imagesPtr.pitch;


    hipPitchedPtr colorPtr = { 0 };
    // cudaXMalloc((void**)&colorData, width * height * batch_size * sizeof(std::array<float, 2>));
    cudaXMalloc3D(&colorPtr, sizeof(std::array<float, 2>), width,
            height, batch_size);
    std::array<float, 2>* colorData = (std::array<float, 2> *) colorPtr.ptr;
    size_t colorPitch = colorPtr.pitch;

    hipPitchedPtr bitVectorPtr = { 0 };
    // cudaXMalloc((void**)&bitVectorData, width * height * batch_size * sizeof(uint8_t));
    cudaXMalloc3D(&bitVectorPtr, sizeof(uint8_t), width,
            height, batch_size);
    size_t bitVecPitch = bitVectorPtr.pitch;
    uint8_t* bitVectorData = (uint8_t *) bitVectorPtr.ptr;

    hipPitchedPtr texturePtr = { 0 };
    // cudaXMalloc((void**)&textureData, width * height * batch_size * sizeof(float));
    cudaXMalloc3D(&texturePtr, sizeof(float), width,
            height, batch_size);
    size_t texturePitch = texturePtr.pitch;
    float* textureData = (float *) texturePtr.ptr;

    hipPitchedPtr batchMasksPtr = { 0 };
    //cudaXMalloc((void**)&batch_masks, width * height * batch_size * sizeof(Bit));
    cudaXMalloc3D(&batchMasksPtr, sizeof(Bit), width,
            height, batch_size);
    size_t masksPitch = batchMasksPtr.pitch;
    Bit* batch_masks = (Bit *) batchMasksPtr.ptr;

    Bit* data_to_save;
    cudaXMallocHost((void **) &data_to_save, width * height * sizeof(Bit) *
                files.size());

    int image_len = files.size();
    //for (auto it = images.begin(); it != images.end(); it += batch_size)
    for (Pixel* batch : batches)
    {
        image_len -= batch_size;
        if (image_len < 0)
            batch_size += image_len;


        // auto batch_end = it + batch_size;
        // if (batch_end > images.end())
        // {
        //     batch_end = images.end();
        //     batch_size = batch_end - it;
        // }



        // for (size_t i = 0; i < batch_size; i++)
           // cudaXMemcpy(imagesData + i * width * height, images[it - images.begin() + i]->get_data().data(), width * height * sizeof(Pixel), hipMemcpyHostToDevice);
        std::cout << "Image pitch: " << imagePitch << '\n';
        std::cout << "batch pitch: " <<  width * sizeof(Pixel) << '\n';
        hipPitchedPtr hostPtr = make_hipPitchedPtr (batch, width *
                sizeof(Pixel), width, height);
        cudaXMemcpy3D(imagesPtr, hostPtr, batch_size, hipMemcpyHostToDevice);

        dim3 blockSize(16, 16, 4);
        dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y, (batch_size + blockSize.z - 1) / blockSize.z);

        calculateSimilarityMeasures<<<gridSize, blockSize, 0,
            stream1>>>(imagesData, backgroundData, colorData, batch_size, width,
                    height, imagePitch, colorPitch);
        calculateBitVector<<<gridSize, blockSize, 0, stream2>>>(imagesData,
                bitVectorData, batch_size, width, height, imagePitch,
                bitVecPitch);
        calculateTextureComponents<<<gridSize, blockSize, 0,
            stream2>>>(bitVectorData, backgroundBitVector, textureData,
                    batch_size, width, height, bitVecPitch, texturePitch);
        hipDeviceSynchronize();

        calculateChoquetMask<<<gridSize, blockSize>>>(colorData, textureData,
                batch_masks, batch_size, width, height, colorPitch, texturePitch, masksPitch);
        hipDeviceSynchronize();


        hipPitchedPtr hostResPtr = make_hipPitchedPtr(data_to_save, width *
                sizeof(Bit), width, height);
        cudaXMemcpy3D(hostResPtr, batchMasksPtr, batch_size, hipMemcpyDeviceToHost);
    }

    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();

    shared_mask mask = std::make_shared<Image<Bit>>(width, height);
    for (size_t i = 0; i < files.size(); i++)
    {
        mask->set_data(data_to_save + i * width * height);
        char nb[6];
        snprintf(nb, 6, "%05lu", i);
        save_mask("dataset/results/mask_" + std::string(nb) + ".png", mask);
    }

    float total_time = std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count();
    std::cout << "Elapsed time: " << total_time << " ms" << std::endl;
    float fps = 1000.0f / (total_time / files.size());
    std::cout << "FPS: " << fps << std::endl;
    std::cout << "PPS: " << fps * width * height << std::endl; 

    cudaXFree(backgroundData);
    cudaXFree(imagesData);
    cudaXFree(colorData);
    cudaXFree(bitVectorData);
    cudaXFree(textureData);
    cudaXFree(batch_masks);
    cudaXFreeHost(data_to_save);

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    hipDeviceReset();

    return EXIT_SUCCESS;
}
