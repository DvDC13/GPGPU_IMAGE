#include "hip/hip_runtime.h"
#include <algorithm>
#include <filesystem>

#include "wrapper_gpu.cuh"
#include "similarityMeasuresC.cuh"
#include "featuresExtractionT.cuh"
#include "similarityMeasuresT.cuh"
#include "choquet.cuh"

#include "image.cuh"

hipStream_t stream1;
hipStream_t stream2;

uint8_t* getBitVector(shared_image image)
{
    int width = image->get_width();
    int height = image->get_height();
    int size = width * height;

    // Allocate device memory
    Pixel* deviceImageData;
    uint8_t* deviceBitVectorData;
    cudaXCalloc((void**)&deviceImageData, size * sizeof(Pixel));
    cudaXCalloc((void**)&deviceBitVectorData, size * sizeof(uint8_t));

    // Copy image data from host to device
    cudaXMemcpy(deviceImageData, image->get_data().data(), size * sizeof(Pixel), hipMemcpyHostToDevice);

    // Launch the kernel
    dim3 blockSize(32, 32);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    calculateBitVectorBackground<<<gridSize, blockSize>>>(deviceImageData, deviceBitVectorData, width, height);
    hipDeviceSynchronize();

    return deviceBitVectorData;
}

int main(int argc, char** argv)
{
    if (argc != 2)
    {
        std::cout << "Usage: " << argv[0] << " <path_to_dataset>" << std::endl;
        return EXIT_FAILURE;
    }

    std::vector<std::string> files;
    std::string path = std::string(argv[1]);

    std::vector<shared_image> images;

    for (const auto& entry : std::filesystem::directory_iterator(path))
        files.push_back(entry.path());

    std::sort(files.begin(), files.end());

    files.reserve(files.size() - 1);

    for (auto it = files.begin() + 1; it != files.end(); it++)
        images.push_back(load_png(*it));

    shared_image background = load_png(files[0]);

    size_t height = images[0]->get_height();
    size_t width = images[0]->get_width();

    size_t memory_usage = width * height * ((sizeof(Pixel)) + sizeof(uint8_t) + sizeof(float) + sizeof(Bit) + sizeof(std::array<float, 2>));
    size_t memory_usage_bg = width * height * (sizeof(uint8_t) + sizeof(Pixel));

    size_t maximum_global_memory = 0;
    hipMemGetInfo(&maximum_global_memory, nullptr);
    size_t max_batch_size = std::floor((maximum_global_memory - memory_usage_bg) / memory_usage);
    size_t batch_size = std::min(max_batch_size, images.size());

    std::cout << "Height: " << height << std::endl;
    std::cout << "Width: " << width << std::endl;
    std::cout << "Memory usage: " << memory_usage << std::endl;
    std::cout << "Memory usage background: " << memory_usage_bg << std::endl;
    std::cout << "Maximum global memory: " << maximum_global_memory << std::endl;
    std::cout << "Maximum batch size: " << max_batch_size << std::endl;
    std::cout << "Batch size: " << batch_size << std::endl;

    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    hipStream_t stream1;
    hipStream_t stream2;

    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    uint8_t* backgroundBitVector = getBitVector(background);

    Pixel* backgroundData;
    cudaXCalloc((void**)&backgroundData, width * height * sizeof(Pixel));
    cudaXMemcpy(backgroundData, background->get_data().data(), width * height * sizeof(Pixel), hipMemcpyHostToDevice);

    Pixel* imagesData;
    cudaXCalloc((void**)&imagesData, width * height * batch_size * sizeof(Pixel));

    std::array<float, 2>* colorData;
    cudaXCalloc((void**)&colorData, width * height * batch_size * sizeof(std::array<float, 2>));

    uint8_t* bitVectorData;
    cudaXCalloc((void**)&bitVectorData, width * height * batch_size * sizeof(uint8_t));

    float* textureData;
    cudaXCalloc((void**)&textureData, width * height * batch_size * sizeof(float));

    Bit* batch_masks;
    cudaXCalloc((void**)&batch_masks, width * height * batch_size * sizeof(Bit));

    Bit* data_to_save;
    cudaXMallocHost((void**)&data_to_save, width * height * images.size() * sizeof(Bit));

    for (auto it = images.begin(); it != images.end(); it += batch_size)
    {
        auto batch_end = it + batch_size;
        if (batch_end > images.end())
        {
            batch_end = images.end();
            batch_size = batch_end - it;
        }

        for (size_t i = 0; i < batch_size; i++)
            cudaXMemcpy(imagesData + i * width * height, images[it - images.begin() + i]->get_data().data(), width * height * sizeof(Pixel), hipMemcpyHostToDevice);

        shared_image* batch_images = &(*it);

        dim3 blockSize(16, 16, 4);
        dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y, (batch_size + blockSize.z - 1) / blockSize.z);

        calculateSimilarityMeasures<<<gridSize, blockSize, 0, stream1>>>(imagesData, backgroundData, colorData, batch_size, width, height);
        calculateBitVector<<<gridSize, blockSize, 0, stream2>>>(imagesData, bitVectorData, batch_size, width, height);
        calculateTextureComponents<<<gridSize, blockSize, 0, stream2>>>(bitVectorData, backgroundBitVector, textureData, batch_size, width, height);
        hipDeviceSynchronize();

        calculateChoquetMask<<<gridSize, blockSize>>>(colorData, textureData, batch_masks, batch_size, width, height);
        hipDeviceSynchronize();

        cudaXMemcpy(data_to_save + (it - images.begin()) * width * height, batch_masks, width * height * batch_size * sizeof(Bit), hipMemcpyDeviceToHost);

        if (hipPeekAtLastError())
            gpuAssert(hipPeekAtLastError(), __FILE__, __LINE__);
    }

    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();

    shared_mask mask = std::make_shared<Image<Bit>>(width, height);
    for (size_t i = 0; i < images.size(); i++)
    {
        mask->set_data(data_to_save + i * width * height);
        char nb[6];
        snprintf(nb, 6, "%05lu", i);
        save_mask("dataset/results/mask_" + std::string(nb) + ".png", mask);
    }

    float total_time = std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count();
    std::cout << "Elapsed time: " << total_time << " ms" << std::endl;
    float fps = 1000.0f / (total_time / images.size());
    std::cout << "FPS: " << fps << std::endl;
    std::cout << "PPS: " << fps * width * height << std::endl; 

    cudaXFree(backgroundData);
    cudaXFree(imagesData);
    cudaXFree(colorData);
    cudaXFree(bitVectorData);
    cudaXFree(textureData);
    cudaXFree(batch_masks);
    cudaXFreeHost(data_to_save);

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    hipDeviceReset();

    return EXIT_SUCCESS;
}
